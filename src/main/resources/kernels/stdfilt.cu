
#include <hip/hip_runtime.h>
extern "C"


__global__ void stdfilt(int *d_Data, int *d_Result, int dataW, int dataH, int kernel_nWidth, int kernel_nHeight)
{
  	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	int ave = 0, ave_cnt = 0, dev = 0, dev_cnt = 0, tmp_dev = 0;
	int inf = 1 / ave;
	int kernel_mi = 0, kernel_mj = 0;
	int offset_x = 0, offset_y = 0;
	int mj = 0, mi = 0;


	//	int tid;

	//	tid = dataW*row + col;

	if (row > dataH || col > dataW)
		return;


	kernel_mi = kernel_nHeight / 2;
	kernel_mj = kernel_nWidth / 2;



	for (mi = kernel_mi*-1; mi <= kernel_mi; mi += 1) {
		for (mj = kernel_mj*-1; mj <= kernel_mj; mj += 1) {
			offset_y = row + mi;                                 //  커널보다 x,y 크기가 작을경우 offset 값은 음수
			offset_x = col + mj;


			if (offset_y < 0 || offset_y >= dataH)     //  y가 커널보다 작거나 크면 음수,  - mi를 더해줌 //  크면 -mi를 더해줌으로써 이미지 범위를 벗어나는 것을 방지
				offset_y = row + (mi*-1);


			if (offset_x < 0 || offset_x >= dataW)
				offset_x = col + (mj*-1);

			//					
			//if (kernel->data[(kernel_mi + mi)*kernel->nWidth + kernel_mj + mj] != 1)      //불필요한 if
			//{
			//	printf("this is first if\n");
			//	continue;
			//}
			//if ((offset_y*dst->nWidth) + offset_x >= src->nHeight*src->nWidth)			 //불필요한 if
			//{ 
			//	printf("this is second if\n");
			//	continue;
			//}


			ave += d_Data[(offset_y*dataW) + offset_x];       /// 필터영역을 모두 더해서 평균값을 구함 편차
			ave_cnt++;


		}
	}

	ave /= ave_cnt;


	for (mi = kernel_mi*-1; mi <= kernel_mi; mi += 1) {                   /// 필터영역을 모두 더해서 평균값을 구함
		for (mj = kernel_mj*-1; mj <= kernel_mj; mj += 1) {
			offset_y = row + mi;
			offset_x = col + mj;
			if (offset_y < 0 || offset_y >= dataH)
				offset_y = row + (mi*-1);
			if (offset_x < 0 || offset_x >= dataW)
				offset_x = col + (mj*-1);

			//if ((offset_y*dst->nWidth) + offset_x >= src->nHeight*src->nWidth)		 //불필요한 if
			//{ 
			//	printf("this is third if");
			//	continue;
			//}

			tmp_dev = (d_Data[(offset_y*dataW) + offset_x] - ave)*(d_Data[(offset_y*dataW) + offset_x] - ave); //편차의 제곱
			if (inf == tmp_dev)
				tmp_dev = 0;

			dev += tmp_dev;
			dev_cnt += 1;
		}
	}


//	tmp_dev = sqrt(dev/ (dev_cnt == kernel_nHeight*kernel_nWidth ? dev_cnt -= 1 : dev_cnt));    //반올림
	if (inf != tmp_dev)
		d_Result[row*dataW + col] = tmp_dev;
	//		src[i*nWidth + j] = ave;
	ave = 0;
	ave_cnt = 0;
	dev = 0;
	dev_cnt = 0;

	//d_Result[tid] = tid;

}