
#include <hip/hip_runtime.h>
extern "C"


__global__ void stdfilt(int *d_Data, int *d_Result, int dataW, int dataH, int kernel_nWidth, int kernel_nHeight)
{
  	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	int ave = 0, ave_cnt = 0, dev = 0, dev_cnt = 0, tmp_dev = 0;
	int inf = 1 / ave;
	int kernel_mi = 0, kernel_mj = 0;
	int offset_x = 0, offset_y = 0;
	int mj = 0, mi = 0;


	//	int tid;

	//	tid = dataW*row + col;

	if (row > dataH || col > dataW)
		return;


	kernel_mi = kernel_nHeight / 2;
	kernel_mj = kernel_nWidth / 2;



	for (mi = kernel_mi*-1; mi <= kernel_mi; mi += 1) {
		for (mj = kernel_mj*-1; mj <= kernel_mj; mj += 1) {
			offset_y = row + mi;                                 //  Ŀ�κ��� x,y ũ�Ⱑ ������� offset ���� ����
			offset_x = col + mj;


			if (offset_y < 0 || offset_y >= dataH)     //  y�� Ŀ�κ��� �۰ų� ũ�� ����,  - mi�� ������ //  ũ�� -mi�� ���������ν� �̹��� ������ ����� ���� ����
				offset_y = row + (mi*-1);


			if (offset_x < 0 || offset_x >= dataW)
				offset_x = col + (mj*-1);

			//					
			//if (kernel->data[(kernel_mi + mi)*kernel->nWidth + kernel_mj + mj] != 1)      //���ʿ��� if
			//{
			//	printf("this is first if\n");
			//	continue;
			//}
			//if ((offset_y*dst->nWidth) + offset_x >= src->nHeight*src->nWidth)			 //���ʿ��� if
			//{ 
			//	printf("this is second if\n");
			//	continue;
			//}


			ave += d_Data[(offset_y*dataW) + offset_x];       /// ���Ϳ����� ��� ���ؼ� ��հ��� ���� ����
			ave_cnt++;


		}
	}

	ave /= ave_cnt;


	for (mi = kernel_mi*-1; mi <= kernel_mi; mi += 1) {                   /// ���Ϳ����� ��� ���ؼ� ��հ��� ����
		for (mj = kernel_mj*-1; mj <= kernel_mj; mj += 1) {
			offset_y = row + mi;
			offset_x = col + mj;
			if (offset_y < 0 || offset_y >= dataH)
				offset_y = row + (mi*-1);
			if (offset_x < 0 || offset_x >= dataW)
				offset_x = col + (mj*-1);

			//if ((offset_y*dst->nWidth) + offset_x >= src->nHeight*src->nWidth)		 //���ʿ��� if
			//{ 
			//	printf("this is third if");
			//	continue;
			//}

			tmp_dev = (d_Data[(offset_y*dataW) + offset_x] - ave)*(d_Data[(offset_y*dataW) + offset_x] - ave); //������ ����
			if (inf == tmp_dev)
				tmp_dev = 0;

			dev += tmp_dev;
			dev_cnt += 1;
		}
	}


//	tmp_dev = sqrt(dev/ (dev_cnt == kernel_nHeight*kernel_nWidth ? dev_cnt -= 1 : dev_cnt));    //�ݿø�
	if (inf != tmp_dev)
		d_Result[row*dataW + col] = tmp_dev;
	//		src[i*nWidth + j] = ave;
	ave = 0;
	ave_cnt = 0;
	dev = 0;
	dev_cnt = 0;

	//d_Result[tid] = tid;

}